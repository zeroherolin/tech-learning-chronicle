
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_gpu(void) {
    printf("GPU: Hello World!\n");
}

int main(void) {
    printf("CPU: Hello World!\n");

    hello_gpu<<<1, 1>>>();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
            printf("CUDA Error: %s\n", hipGetErrorString(err));
    }

    hipDeviceSynchronize(); // 等待GPU完成

    return 0;
}

